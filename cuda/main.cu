
#include <hip/hip_runtime.h>
#include<stdio.h> 
#include<stdlib.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Kernel
__global__ void convert_to_gray(unsigned char* color_array, unsigned char* gray_array, int x, int y)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;

	if(id < x*y) gray_array[id] = color_array[3*(id)+0]*0.21 + color_array[3*(id)+1]*0.72 + color_array[3*(id)+2]*0.07;
	
}	

// Main program
int main() 
{ 
	// Open input file
//	FILE* in_file = fopen("original_file.ppm","rb"); 
	FILE* in_file = fopen("hackathon.ppm","rb");
	if (in_file==NULL){ 
		printf("File does not exist.");
		return 0;
	} 

	// Read file format, # of pixels in x and y dim, max value for each channel
	char fmt[10];
	char max[10];
	int dimx;
	int dimy;

	fscanf(in_file, "%s", fmt);
	fscanf(in_file, "%d", &dimx);
	fscanf(in_file, "%d", &dimy);
	fscanf(in_file, "%s%*[\n]", max);

	// Allocate memory for array to hold RGB values for all pixels
	unsigned char *rgb_image = (unsigned char*)malloc(dimx*dimy*3*sizeof(unsigned char));

	// Read in pixel data from input file
	fread(rgb_image, dimx*dimy*3, sizeof(unsigned char), in_file);
	fclose(in_file);

	/* Calculate grayscale values based on RGB values and write output file --- */

	// Allocate memory for array to hold grayscale values for all pixels
	unsigned char *gray_image = (unsigned char*)malloc(dimx*dimy*sizeof(unsigned char));

	// Allocate GPU memory
	unsigned char *d_rgb_image, *d_gray_image;
	cudaErrorCheck( hipMalloc(&d_rgb_image, dimx*dimy*3*sizeof(unsigned char)) );
	cudaErrorCheck( hipMalloc(&d_gray_image, dimx*dimy*sizeof(unsigned char)) );

	cudaErrorCheck( hipMemcpy(d_rgb_image, rgb_image, dimx*dimy*3*sizeof(unsigned char), hipMemcpyHostToDevice) );

  // Set execution configuration parameters
  //    thr_per_blk: number of CUDA threads per grid block
  //    blk_in_grid: number of blocks in grid
  int thr_per_blk = 256;
  int blk_in_grid = ceil( float(dimx*dimy) / thr_per_blk );

  // Launch kernel
  convert_to_gray<<< blk_in_grid, thr_per_blk >>>(d_rgb_image, d_gray_image, dimx, dimy);

  // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
  hipError_t cuErrSync  = hipGetLastError();

  // Check for errors on the GPU after control is returned to CPU
  hipError_t cuErrAsync = hipDeviceSynchronize();

  if (cuErrSync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync)); exit(0); }
  if (cuErrAsync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync)); exit(0); }

	cudaErrorCheck( hipMemcpy(gray_image, d_gray_image, dimx*dimy, hipMemcpyDeviceToHost) );

  // Write pixels to PGM P5 formatted file
  FILE *out_file = fopen("output.pgm", "wb");
  fprintf(out_file, "P5\n%d %d\n%d\n", dimx, dimy, 255);
  fwrite(gray_image, sizeof(unsigned char), dimx*dimy, out_file);
  fclose(out_file);

	cudaErrorCheck( hipFree(d_rgb_image) );
	cudaErrorCheck( hipFree(d_gray_image) );

	free(rgb_image);
	free(gray_image);
 
  printf("\n---------------------------\n");
  printf("__SUCCESS__\n");
  printf("---------------------------\n");
  printf("dimx              = %d\n", dimx);
  printf("dimy              = %d\n", dimy);
  printf("Threads Per Block = %d\n", thr_per_blk);
  printf("Blocks In Grid    = %d\n", blk_in_grid);
  printf("---------------------------\n\n");
 
	return 0; 
} 
